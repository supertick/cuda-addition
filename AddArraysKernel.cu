
#include <hip/hip_runtime.h>
extern "C" __global__ void addArrays(const float *a, const float *b, float *c, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        c[i] = a[i] + b[i];
    }
}
